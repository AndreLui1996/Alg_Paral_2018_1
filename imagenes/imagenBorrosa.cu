#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>

using namespace std;

#define CHANNELS 3
__global__ 
void blurKernel(unsigned char* in, unsigned char* out, int w, int h) {
	int Col  = blockIdx.x * blockDim.x + threadIdx.x;
	int Row  = blockIdx.y * blockDim.y + threadIdx.y;

	if (Col < w && Row < h) {
		int pixVal = 0;
		int pixels = 0;

		// Get the average of the surrounding 2xBLUR_SIZE x 2xBLUR_SIZE box
		for(int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE+1; ++blurRow) {
			for(int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE+1; ++blurCol) {

				int curRow = Row + blurRow;
				int curCol = Col + blurCol;
				// Verify we have a valid image pixel
				if(curRow > -1 && curRow < h && curCol > -1 && curCol < w) {
					pixVal += in[curRow * w + curCol];
					pixels++; // Keep track of number of pixels in the accumulated total
				}
			}
		}

		// Write our new pixel value out
		out[Row * w + Col] = (unsigned char)(pixVal / pixels);
	}
}

int main(int argc, char** argv){
	
	if (argc < 3){
		cout << "Usage: " << argv[0] << " <input.png>" << " <output.png>" << endl;
		exit(1);
	}
	
	string input_file;
	string output_cuda_file  = "../input.png";
	
	input_file  = string(argv[1]);
	output_cuda_file = string(argv[2]);



	return 0;

}
